#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unordered_map>
#include <iostream>
#include <time.h>
//#include <gl/glew.h>
////#include <gl/GL.h>
//#include <gl/glut.h>
#include <helper_gl.h>
#include <GL/freeglut.h>

//#include "hip/hip_runtime.h"
//#include "cuda_gl_interop.h"
//#include "device_atomic_functions.h"
//#include ""
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <helper_cuda_gl.h>      // helper functions for CUDA/GL interop
#include "math_functions.h"

#include "KMath.h"
#include "RayTracer.h"
#include "Camera.h"
#include "PointLight.h"
#include "Object.h"
#include "Sphere.h"
#include "Triangle.h"
#include "Plane.h"

//#pragma comment(lib, "glut32.lib")
//#pragma comment(lib, "glu32.lib")
//#pragma comment(lib, "opengl32.lib")
//#pragma comment(lib, "glut32.lib")
//#pragma comment(lib, "glew32.lib")
//#pragma comment(lib, "glew32s.lib")

using namespace std;

// �ȼ� ���۸� ����Ű�� OpenGL �ڵ� 
GLuint gl_pbo;
// �ȼ� ���۸� ����Ű�� CUDA �ڵ� 
struct hipGraphicsResource *cuda_pbo_resource;
// �ȼ� ���ۿ� ���� ���� �޸� �ּ�
uchar4 *dev_data;
// RayTracer
RayTracer* ray_tracer;
const int width = 1600;
const int height = 900;
// Camera
KPoint3 camera_pos;
KVector3 camera_up;
KVector3 camera_look;
float camera_fovx;
Camera camera;
// Objects
std::unordered_map<int, Object*> objects;
std::unordered_map<int, PointLight*> lights;

// OpenGL
bool is_camera_rotate = false;
bool is_object_select = false;
GLint beforePoint[2];
int selected_object = -1;
KPoint3 selected_object_pos;

int id;
Camera* dev_camera;
KVector3* dev_kvec, *dev_kvec2;
KPoint3* dev_kpos;
Color* dev_diffuse, *dev_specular;
float* dev_fovx;
int* dev_id;

// Scene Functions
void scene_set_camera(__in const KPoint3& pos, __in const KVector3& up, 
					__in const KVector3& look, __in const float& fovx);
void scene_add_sphere(__in const KPoint3& pos, __in const Color& diffuse, __in const Color& specular,
					__in const float& r, __in const float& shininess, __in const float& reflect,
					__in const float& refract, __in const float& density, __out int& id);
void scene_add_plane(__in const KVector3& normal, __in const KPoint3& point,
					__in const Color& diffuse, __in const Color& specular, 
					__in const float& shininess, __in const float& reflect,
					__in const float& refract, __in const float& density, __out int& id);
void scene_add_mesh(__in const char* filename,
					__in const KPoint3& point, __in const Color& diffuse,
					__in const Color& specular, __in const float& shininess,
					__in const float& reflect, __in const float& refract,
					__in const float& density, __out int& id);
void scene_modify_sphere(__in int id, __in const KPoint3& pos, __in const Color& diffuse, __in const Color& specular,
						__in const float& r, __in const float& shininess, __in const float& reflect,
						__in const float& refract, __in const float& density);
void scene_modify_plane(__in int id, __in const KVector3& normal, __in const KPoint3& point,
						__in const Color& diffuse, __in const Color& specular,
						__in const float& shininess, __in const float& reflect,
						__in const float& refract, __in const float& density);
void scene_modify_mesh(__in int id, __in const KPoint3& point, __in const Color& diffuse,
						__in const Color& specular, __in const float& shininess,
						__in const float& reflect, __in const float& refract,
						__in const float& density);
void scene_add_point_light(__in const KPoint3& point, __in const Color& color, __out int& id);
void scene_modify_point_light(__in int id, __in const KPoint3& point, __in const Color& color);

// OPENGL Functions
void mouse(__in int button, __in int state, __in int x, __in int y);
void motion(__in int _x, __in int _y);

void destroy_buffer(GLuint* buffer)
{
	glBindBuffer(GL_TEXTURE_2D, 0);
	glDeleteBuffers(1, buffer);
	*buffer = 0;
}

void display_func()
{
	glClearColor(0, 1, 1, 1);
	glClear(GL_COLOR_BUFFER_BIT);

	// Ŀ���� ���� ����̽� �޸��� ���� �ּ�(dev_data)�� ����
	size_t size;
	hipGraphicsMapResources(1, &cuda_pbo_resource, NULL);
	hipGraphicsResourceGetMappedPointer((void **)&dev_data, &size, cuda_pbo_resource);
	
	dim3 dimGrim(50, 30);
	dim3 dimBlock(width / dimGrim.x, height / dimGrim.y);
	clock_t time_st = clock();
	RayTrace << <dimGrim, dimBlock >> > (ray_tracer, (unsigned char*)dev_data);

	// �����ڿ��� ���� ����
	hipGraphicsUnmapResources(1, &cuda_pbo_resource, NULL);

	hipDeviceSynchronize();

	glDrawPixels(width, height, GL_RGBA, GL_UNSIGNED_BYTE, 0);

	clock_t time_ed = clock();
	cout << "Elapsed time: " << time_ed - time_st << "ms" << endl;

	//glFinish();
	glutSwapBuffers();
}

int main(int argc, char* argv[])
{
	// GLUT �ʱ�ȭ
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_RGBA);

	// ������ ũ�� ���� �� ����
	glutInitWindowSize(width, height);
	glutCreateWindow("Cuda RayTracer");

	// �ݹ� �Լ� ���
	glutDisplayFunc(display_func);
	glutMouseFunc(mouse);
	glutMotionFunc(motion);

	// OpenGL 2.0���� ���θ� ����
	if (!isGLVersionSupported(2, 0))
		return 0;	

	hipSetDevice(0);

	// ������ �ڵ��� �����ϰ�, �ڵ��� �ȼ� ���ۿ� ���ε�
	glGenBuffers(1, &gl_pbo);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, gl_pbo);

	// OpenGL�� �ȼ� ���� �Ҵ� ��û
	// �Ķ���� ����:
	//		NULL: �ʱ�ȭ ������ ����
	//		GL_DYNAMIC_DRAW_ARB: ���۰� �ݺ������� ������ ����	
	glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, width * height * 4, NULL, GL_DYNAMIC_DRAW_ARB);

	// OpenGL�� �ȼ� ���۸� �׷��� ���ҽ��μ� CUDA �ý��۰� ������ ���� �˸�
	// ���� cuda_pbo_resource�� CUDA���� ����� �ȼ� ���ۿ� ���� �ڵ� ���� ����
	// ���: cudaGraphicsMapFlagsNone, cudaGraphicsmapFlagsReadOnly, cudaGraphicsMapFlagsWriteDiscard, ��
	hipGraphicsGLRegisterBuffer(&cuda_pbo_resource, gl_pbo, cudaGraphicsMapFlagsNone);

	// Ŀ���� ���� ����̽� �޸��� ���� �ּ�(dev_data)�� ����
	size_t size;
	hipGraphicsMapResources(1, &cuda_pbo_resource, NULL);
	hipGraphicsResourceGetMappedPointer((void **)&dev_data, &size, cuda_pbo_resource);

	// ��� ����, �� ������ ����
	hipDeviceSetLimit(hipLimitStackSize, 10000);
	hipDeviceSetLimit(hipLimitMallocHeapSize, width * height * sizeof(Color) * 5);

	hipMalloc((void**)&ray_tracer, sizeof(RayTracer));
	hipMalloc((void**)&dev_camera, sizeof(Camera));
	hipMalloc((void**)&dev_kvec, sizeof(KVector3));
	hipMalloc((void**)&dev_kvec2, sizeof(KVector3));
	hipMalloc((void**)&dev_kpos, sizeof(KPoint3));
	hipMalloc((void**)&dev_fovx, sizeof(float));
	hipMalloc((void**)&dev_diffuse, sizeof(Color));
	hipMalloc((void**)&dev_specular, sizeof(Color));
	hipMalloc((void**)&dev_id, sizeof(int));

	// Camera
	camera_pos = KPoint3(0.0f, 0.0f, -60.0f);
	camera_up = KVector3(0.0f, 1.0f, 0.0f);
	camera_look = KVector3(0.0f, 0.0f, 1.0f);
	camera_fovx = 120.0f;
	scene_set_camera(camera_pos, camera_up, camera_look, camera_fovx);

	// Light
	scene_add_point_light(KPoint3(0.0f, 15.0f, 0.0f), Color(200, 200, 200), id);

	// �� 25��
	for(int i = 0; i < 2; i++)
		for (int j = 0; j < 3; j++)
		{
			scene_add_sphere(KPoint3(i * 7.0f - 6.0f, 2.0f, j * 7.0f - 6.0f), Color(i * 30 + 100, j * 30 + 100, 0),
							Color(200, 200, 200), 1.5f, 20.0f, 0.35f, 0.0f, 1.2f, id);
		}

	// �������� �Ѱ�
	scene_add_sphere(KPoint3(10.0f, 8.0f, -10.0f), Color(0, 0, 0),
					Color(0, 0, 0), 3.0f, 20.0f, 0.0f, 1.0f, 1.58f, id);

	// Plane
	scene_add_plane(KVector3(0.0f, 1.0f, 0.0f), KPoint3(0.0f, -4.0f, 0.0f),
					Color(140, 140, 140), Color(140, 140, 140), 30.0f, 0.0f, 0.0f, 1.2f, id);

	SetImageResolution<<<1, 1>>>(ray_tracer, width, height);

	// �̺�Ʈ ó�� ���� ����
	glutMainLoop();
	
	// delete
	hipGraphicsUnregisterResource(cuda_pbo_resource);
	destroy_buffer(&gl_pbo);
	hipFree(ray_tracer);
	hipFree(dev_camera);
	hipFree(dev_diffuse);
	hipFree(dev_specular);
	hipFree(dev_id);
	hipFree(dev_kpos);
	hipFree(dev_kvec);
	hipFree(dev_kvec2);
	hipFree(dev_fovx);

	return 0;
}

void scene_set_camera(const KPoint3& pos, const KVector3& up,
					const KVector3& look, const float& fovx)
{
	// Camera
	hipMemcpy(dev_kpos, &pos, sizeof(KPoint3), hipMemcpyHostToDevice);
	hipMemcpy(dev_kvec, &up, sizeof(KVector3), hipMemcpyHostToDevice);
	hipMemcpy(dev_kvec2, &look, sizeof(KVector3), hipMemcpyHostToDevice);
	hipMemcpy(dev_fovx, &fovx, sizeof(float), hipMemcpyHostToDevice);
	SetCamera << <1, 1 >> >(ray_tracer, dev_kpos, dev_kvec, dev_kvec2, dev_fovx);
}

void scene_add_sphere(const KPoint3& pos, const Color& diffuse, const Color& specular,
					const float& r, const float& shininess, const float& reflect,
					const float& refract, const float& density, int& id)
{
	// gpu
	hipMemcpy(dev_kpos, &pos, sizeof(KPoint3), hipMemcpyHostToDevice);
	hipMemcpy(dev_diffuse, &diffuse, sizeof(Color), hipMemcpyHostToDevice);
	hipMemcpy(dev_specular, &specular, sizeof(Color), hipMemcpyHostToDevice);
	AddSphere << <1, 1 >> >(ray_tracer, dev_kpos, r, dev_diffuse, dev_specular, shininess, reflect, refract, density, dev_id);
	hipMemcpy(&id, dev_id, sizeof(int), hipMemcpyDeviceToHost);
	// cpu
	Object* object_ptr = new Sphere(pos, r, diffuse, specular, shininess, reflect, refract, density);
	objects.insert(pair<int, Object*>(id, object_ptr));
}

void scene_add_plane(const KVector3& normal, const KPoint3& point,
					const Color& diffuse, const Color& specular,
					const float& shininess, const float& reflect,
					const float& refract, const float& density, int& id)
{
	// gpu
	hipMemcpy(dev_kvec, &normal, sizeof(KVector3), hipMemcpyHostToDevice);
	hipMemcpy(dev_kpos, &point, sizeof(KPoint3), hipMemcpyHostToDevice);
	hipMemcpy(dev_diffuse, &diffuse, sizeof(Color), hipMemcpyHostToDevice);
	hipMemcpy(dev_specular, &specular, sizeof(Color), hipMemcpyHostToDevice);
	AddPlane << <1, 1 >> >(ray_tracer, dev_kvec, dev_kpos, dev_diffuse, dev_specular, shininess, reflect, refract, density, dev_id);
	hipMemcpy(&id, dev_id, sizeof(int), hipMemcpyDeviceToHost);
	// cpu
	Object* object_ptr = new Plane(normal, point, diffuse, specular, shininess, reflect, refract, density);
	objects.insert(pair<int, Object*>(id, object_ptr));
}

void scene_add_mesh(__in const char* filename,
	__in const KPoint3& point, __in const Color& diffuse,
	__in const Color& specular, __in const float& shininess,
	__in const float& reflect, __in const float& refract,
	__in const float& density, __out int& id)
{
	// gpu
	hipMemcpy(dev_kpos, &point, sizeof(KPoint3), hipMemcpyHostToDevice);
	hipMemcpy(dev_diffuse, &diffuse, sizeof(Color), hipMemcpyHostToDevice);
	hipMemcpy(dev_specular, &specular, sizeof(Color), hipMemcpyHostToDevice);
	// obj load

}

void scene_modify_sphere(__in int id, __in const KPoint3& pos, __in const Color& diffuse, __in const Color& specular,
						__in const float& r, __in const float& shininess, __in const float& reflect,
						__in const float& refract, __in const float& density)
{
	// gpu
	hipMemcpy(dev_kpos, &pos, sizeof(KPoint3), hipMemcpyHostToDevice);
	hipMemcpy(dev_diffuse, &diffuse, sizeof(Color), hipMemcpyHostToDevice);
	hipMemcpy(dev_specular, &specular, sizeof(Color), hipMemcpyHostToDevice);
	ModifySphere<<<1, 1>>>(id, ray_tracer, dev_kpos, r, dev_diffuse, dev_specular, shininess, reflect, refract, density);
	// cpu
	if (objects[id]->GetType() != SPHERE_TYPE)
		return;
	else
	{
		*objects[id] = Sphere(pos, r, diffuse, specular, shininess, reflect, refract, density);
	}
}

void scene_modify_plane(__in int id, __in const KVector3& normal, __in const KPoint3& point,
						__in const Color& diffuse, __in const Color& specular,
						__in const float& shininess, __in const float& reflect,
						__in const float& refract, __in const float& density)
{
	// gpu
	hipMemcpy(dev_kvec, &normal, sizeof(KVector3), hipMemcpyHostToDevice);
	hipMemcpy(dev_kpos, &point, sizeof(KPoint3), hipMemcpyHostToDevice);
	hipMemcpy(dev_diffuse, &diffuse, sizeof(Color), hipMemcpyHostToDevice);
	hipMemcpy(dev_specular, &specular, sizeof(Color), hipMemcpyHostToDevice);
	ModifyPlane << <1, 1 >> > (id, ray_tracer, dev_kvec, dev_kpos, dev_diffuse, dev_specular, shininess, reflect, refract, density);
	// cpu
	if (objects[id]->GetType() != PLANE_TYPE)
		return;
	else
	{
		*objects[id] = Plane(normal, point, diffuse, specular, shininess, reflect, refract, density);
	}
}

void scene_modify_mesh(__in int id, __in const KPoint3& point, __in const Color& diffuse,
						__in const Color& specular, __in const float& shininess,
						__in const float& reflect, __in const float& refract,
						__in const float& density)
{}

void scene_add_point_light(__in const KPoint3& point, __in const Color& color, __out int& id)
{
	// gpu
	hipMemcpy(dev_kpos, &point, sizeof(KPoint3), hipMemcpyHostToDevice);
	hipMemcpy(dev_diffuse, &color, sizeof(Color), hipMemcpyHostToDevice);
	AddPointLight << <1, 1 >> > (ray_tracer, dev_kpos, dev_diffuse, dev_id);
	hipMemcpy(&id, dev_id, sizeof(int), hipMemcpyDeviceToHost);
	// cpu
	PointLight* light_ptr = new PointLight(point, color);
	lights.insert(pair<int, PointLight*>(id, light_ptr));
}

void scene_modify_point_light(__in int id, __in const KPoint3& point, __in const Color& color)
{
	// gpu
	hipMemcpy(dev_kpos, &point, sizeof(KPoint3), hipMemcpyHostToDevice);
	hipMemcpy(dev_diffuse, &color, sizeof(Color), hipMemcpyHostToDevice);
	ModifyPointLight << <1, 1 >> > (id, ray_tracer, dev_kpos, dev_diffuse);
	// cpu
	if (lights[id]->GetType() != POINT_LIGHT_TYPE)
		return;
	else
	{
		*lights[id] = PointLight(point, color);
	}
}

// OPENGL FUNC
void mouse(__in int button, __in int state, __in int x, __in int y)
{
	// Todo :  ȸ��, �̵�, ũ�� ��ȯ�� ���� ���콺 �κ�
	if (button == GLUT_RIGHT_BUTTON && state == GLUT_DOWN)
	{
		is_camera_rotate = true;
		beforePoint[0] = x;
		beforePoint[1] = y;
	}
	if (button == GLUT_LEFT_BUTTON && state == GLUT_DOWN)
	{
		is_object_select = true;
		GetClickedObj << <1, 1 >> > (ray_tracer, x, y, dev_id);
		hipMemcpy(&selected_object, dev_id, sizeof(int), hipMemcpyDeviceToHost);
		printf("selected id: %d\n", selected_object);
		if (selected_object < 0)
		{
			is_object_select = false;
			return;
		}
		else
		{
			GetObjectPosition << <1, 1 >> > (ray_tracer, selected_object, dev_kpos);
			hipMemcpy(&selected_object_pos, dev_kpos, sizeof(KPoint3), hipMemcpyDeviceToHost);
		}
	}
	
	if (button == GLUT_RIGHT_BUTTON && state == GLUT_UP)
		is_camera_rotate = false;
	if (button == GLUT_LEFT_BUTTON && state == GLUT_UP)
		is_object_select = false;
}
void motion(__in int _x, __in int _y)
{
	// ���� Ŭ���ϰ� �巡�� �� ������Ʈ �̵�
	if (is_object_select)
	{
		KVector3 o = camera.GetScreenO(width, height);
		KPoint3 pos = camera.GetEyePosition();
		KVector3 dir = o +
			_x * camera.GetScreenU() +
			_y * camera.GetScreenV();
		dir = dir.Normalize();
		Ray ray(pos, dir);

		//printf("selected type: %d\n", objects[selected_object]->GetType());

		// �� �����̱�
		if (objects[selected_object]->GetType() == SPHERE_TYPE)
		{
			KPoint3 temp_pos = KPoint3(pos + dir * 60.0f);
			//printf("position: %f, %f, %f\n", temp_pos[0], temp_pos[1], temp_pos[2]);
			scene_modify_sphere(selected_object, temp_pos, objects[selected_object]->GetDiffuse(), objects[selected_object]->GetSpecular(),
								((Sphere*)objects[selected_object])->GetR(), objects[selected_object]->GetShininess(), objects[selected_object]->GetReflectance(),
								objects[selected_object]->GetTransmittance(), objects[selected_object]->GetDensity());
		}

		glutPostRedisplay();
	}

	// ������ Ŭ���ϰ� �巡�� �� ī�޶� ȸ��
	if (is_camera_rotate)
	{
		KVector3 l, r, u;
		KPoint3 newEye;
		GLfloat eyeToNewEye[2];
	
		eyeToNewEye[0] = (_x - beforePoint[0]);
		eyeToNewEye[1] = (_y - beforePoint[1]);
	
		for (int i = 0; i < 3; i++)
			l[i] = -camera_pos[i] / 60.0f;
	
		r[0] = -l[2];
		r[1] = 0;
		r[2] = l[0];
	
		u[0] = -l[0] * l[1];
		u[1] = l[2] * l[2] + l[0] * l[0];
		u[2] = -l[1] * l[2];
	
		for (int i = 0; i < 3; i++)
			newEye[i] = camera_pos[i] + r[i] * 0.2f * (-eyeToNewEye[0]) + u[i] * 0.2f * eyeToNewEye[1];
	
		GLfloat newEyeLength = sqrtf(newEye[0] * newEye[0] + newEye[1] * newEye[1] + newEye[2] * newEye[2]);
	
		for (int i = 0; i < 3; i++)
			newEye[i] = 60.0f * newEye[i] / newEyeLength;
	
		camera_pos = newEye;
	
		camera_look[0] = -camera_pos[0];
		camera_look[1] = -camera_pos[1];
		camera_look[2] = -camera_pos[2];
	
		beforePoint[0] = _x;
		beforePoint[1] = _y;

		camera = Camera(camera_pos, camera_up, camera_look, camera_fovx);
		scene_set_camera(camera_pos, camera_up, camera_look, camera_fovx);
		glutPostRedisplay();
	}
}
